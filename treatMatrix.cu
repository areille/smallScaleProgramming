#include "hip/hip_runtime.h"
/*
*   This program reads a matrix with Matrix Market format,
*   then it turns it into 2 objects :
*    - one matrix in CSR format
*    - one matrix in ELLPACK format
*   Then it proceeds the mulctiplication of this matrix with a vector using cuda
*/

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <string.h>
#include <iostream>
#include <string.h>
#include <ctype.h>

#include <hip/hip_runtime.h> // For CUDA runtime API
#include <hip/hip_runtime_api.h>  // For checkCudaError macro
#include <helper_timer.h> // For CUDA SDK timers

//Simple dimension: define a 1D block structure
#define BD 256

const dim3 BLOCK_DIM(BD);

#define MM_MAX_LINE_LENGTH 1025
#define MatrixMarketBanner "%%MatrixMarket"
#define MM_MAX_TOKEN_LENGTH 64

typedef char MM_typecode[4];
/********************* MM_typecode query fucntions ***************************/

#define mm_is_matrix(typecode) ((typecode)[0] == 'M')

#define mm_is_sparse(typecode) ((typecode)[1] == 'C')
#define mm_is_coordinate(typecode) ((typecode)[1] == 'C')
#define mm_is_dense(typecode) ((typecode)[1] == 'A')
#define mm_is_array(typecode) ((typecode)[1] == 'A')

#define mm_is_complex(typecode) ((typecode)[2] == 'C')
#define mm_is_real(typecode) ((typecode)[2] == 'R')
#define mm_is_pattern(typecode) ((typecode)[2] == 'P')
#define mm_is_integer(typecode) ((typecode)[2] == 'I')

#define mm_is_symmetric(typecode) ((typecode)[3] == 'S')
#define mm_is_general(typecode) ((typecode)[3] == 'G')
#define mm_is_skew(typecode) ((typecode)[3] == 'K')
#define mm_is_hermitian(typecode) ((typecode)[3] == 'H')

int mm_is_valid(MM_typecode matcode); /* too complex for a macro */

/********************* MM_typecode modify fucntions ***************************/

#define mm_set_matrix(typecode) ((*typecode)[0] = 'M')
#define mm_set_coordinate(typecode) ((*typecode)[1] = 'C')
#define mm_set_array(typecode) ((*typecode)[1] = 'A')
#define mm_set_dense(typecode) mm_set_array(typecode)
#define mm_set_sparse(typecode) mm_set_coordinate(typecode)

#define mm_set_complex(typecode) ((*typecode)[2] = 'C')
#define mm_set_real(typecode) ((*typecode)[2] = 'R')
#define mm_set_pattern(typecode) ((*typecode)[2] = 'P')
#define mm_set_integer(typecode) ((*typecode)[2] = 'I')

#define mm_set_symmetric(typecode) ((*typecode)[3] = 'S')
#define mm_set_general(typecode) ((*typecode)[3] = 'G')
#define mm_set_skew(typecode) ((*typecode)[3] = 'K')
#define mm_set_hermitian(typecode) ((*typecode)[3] = 'H')

#define mm_clear_typecode(typecode) ((*typecode)[0] = (*typecode)[1] = \
                                         (*typecode)[2] = ' ',         \
                                     (*typecode)[3] = 'G')

#define mm_initialize_typecode(typecode) mm_clear_typecode(typecode)

/********************* Matrix Market error codes ***************************/

#define MM_COULD_NOT_READ_FILE 11
#define MM_PREMATURE_EOF 12
#define MM_NOT_MTX 13
#define MM_NO_HEADER 14
#define MM_UNSUPPORTED_TYPE 15
#define MM_LINE_TOO_LONG 16
#define MM_COULD_NOT_WRITE_FILE 17

/******************** Matrix Market internal definitions ********************

   MM_matrix_typecode: 4-character sequence

				    ojbect 		sparse/   	data        storage 
						  		dense     	type        scheme

   string position:	 [0]        [1]			[2]         [3]

   Matrix typecode:  M(atrix)  C(oord)		R(eal)   	G(eneral)
						        A(array)	C(omplex)   H(ermitian)
											P(attern)   S(ymmetric)
								    		I(nteger)	K(kew)

 ***********************************************************************/

#define MM_MTX_STR "matrix"
#define MM_ARRAY_STR "array"
#define MM_DENSE_STR "array"
#define MM_COORDINATE_STR "coordinate"
#define MM_SPARSE_STR "coordinate"
#define MM_COMPLEX_STR "complex"
#define MM_REAL_STR "real"
#define MM_INT_STR "integer"
#define MM_GENERAL_STR "general"
#define MM_SYMM_STR "symmetric"
#define MM_HERM_STR "hermitian"
#define MM_SKEW_STR "skew-symmetric"
#define MM_PATTERN_STR "pattern"

int mm_read_mtx_crd_size(FILE *f, int *M, int *N, int *nz)
{
    char line[MM_MAX_LINE_LENGTH];
    int num_items_read;

    /* set return null parameter values, in case we exit with errors */
    *M = *N = *nz = 0;

    /* now continue scanning until you reach the end-of-comments */
    do
    {
        if (fgets(line, MM_MAX_LINE_LENGTH, f) == NULL)
            return MM_PREMATURE_EOF;
    } while (line[0] == '%');

    /* line[] is either blank or has M,N, nz */
    if (sscanf(line, "%d %d %d", M, N, nz) == 3)
        return 0;

    else
        do
        {
            num_items_read = fscanf(f, "%d %d %d", M, N, nz);
            if (num_items_read == EOF)
                return MM_PREMATURE_EOF;
        } while (num_items_read != 3);

    return 0;
}

char *mm_typecode_to_str(MM_typecode matcode)
{
    char buffer[MM_MAX_LINE_LENGTH];
    char *types[4];
    char *mm_strdup(const char *);
    int error = 0;

    /* check for MTX type */
    if (mm_is_matrix(matcode))
        types[0] = MM_MTX_STR;
    else
        error = 1;

    /* check for CRD or ARR matrix */
    if (mm_is_sparse(matcode))
        types[1] = MM_SPARSE_STR;
    else if (mm_is_dense(matcode))
        types[1] = MM_DENSE_STR;
    else
        return NULL;

    /* check for element data type */
    if (mm_is_real(matcode))
        types[2] = MM_REAL_STR;
    else if (mm_is_complex(matcode))
        types[2] = MM_COMPLEX_STR;
    else if (mm_is_pattern(matcode))
        types[2] = MM_PATTERN_STR;
    else if (mm_is_integer(matcode))
        types[2] = MM_INT_STR;
    else
        return NULL;

    /* check for symmetry type */
    if (mm_is_general(matcode))
        types[3] = MM_GENERAL_STR;
    else if (mm_is_symmetric(matcode))
        types[3] = MM_SYMM_STR;
    else if (mm_is_hermitian(matcode))
        types[3] = MM_HERM_STR;
    else if (mm_is_skew(matcode))
        types[3] = MM_SKEW_STR;
    else
        return NULL;

    sprintf(buffer, "%s %s %s %s", types[0], types[1], types[2], types[3]);
    return mm_strdup(buffer);
}

int mm_read_banner(FILE *f, MM_typecode *matcode)
{
    char line[MM_MAX_LINE_LENGTH];
    char banner[MM_MAX_TOKEN_LENGTH];
    char mtx[MM_MAX_TOKEN_LENGTH];
    char crd[MM_MAX_TOKEN_LENGTH];
    char data_type[MM_MAX_TOKEN_LENGTH];
    char storage_scheme[MM_MAX_TOKEN_LENGTH];
    char *p;

    mm_clear_typecode(matcode);

    if (fgets(line, MM_MAX_LINE_LENGTH, f) == NULL)
        return MM_PREMATURE_EOF;

    if (sscanf(line, "%s %s %s %s %s", banner, mtx, crd, data_type,
               storage_scheme) != 5)
        return MM_PREMATURE_EOF;

    for (p = mtx; *p != '\0'; *p = tolower(*p), p++)
        ; /* convert to lower case */
    for (p = crd; *p != '\0'; *p = tolower(*p), p++)
        ;
    for (p = data_type; *p != '\0'; *p = tolower(*p), p++)
        ;
    for (p = storage_scheme; *p != '\0'; *p = tolower(*p), p++)
        ;

    /* check for banner */
    if (strncmp(banner, MatrixMarketBanner, strlen(MatrixMarketBanner)) != 0)
        return MM_NO_HEADER;

    /* first field should be "mtx" */
    if (strcmp(mtx, MM_MTX_STR) != 0)
        return MM_UNSUPPORTED_TYPE;
    mm_set_matrix(matcode);

    /* second field describes whether this is a sparse matrix (in coordinate
storgae) or a dense array */

    if (strcmp(crd, MM_SPARSE_STR) == 0)
        mm_set_sparse(matcode);
    else if (strcmp(crd, MM_DENSE_STR) == 0)
        mm_set_dense(matcode);
    else
        return MM_UNSUPPORTED_TYPE;

    /* third field */

    if (strcmp(data_type, MM_REAL_STR) == 0)
        mm_set_real(matcode);
    else if (strcmp(data_type, MM_COMPLEX_STR) == 0)
        mm_set_complex(matcode);
    else if (strcmp(data_type, MM_PATTERN_STR) == 0)
        mm_set_pattern(matcode);
    else if (strcmp(data_type, MM_INT_STR) == 0)
        mm_set_integer(matcode);
    else
        return MM_UNSUPPORTED_TYPE;

    /* fourth field */

    if (strcmp(storage_scheme, MM_GENERAL_STR) == 0)
        mm_set_general(matcode);
    else if (strcmp(storage_scheme, MM_SYMM_STR) == 0)
        mm_set_symmetric(matcode);
    else if (strcmp(storage_scheme, MM_HERM_STR) == 0)
        mm_set_hermitian(matcode);
    else if (strcmp(storage_scheme, MM_SKEW_STR) == 0)
        mm_set_skew(matcode);
    else
        return MM_UNSUPPORTED_TYPE;

    return 0;
}

int mm_read_unsymmetric_sparse(const char *fname, int *M_, int *N_, int *nz_,
                               double **val_, int **I_, int **J_)
{
    FILE *f;
    MM_typecode matcode;
    int M, N, nz;
    int i;
    double *val;
    int *I, *J;

    if ((f = fopen(fname, "r")) == NULL)
        return -1;

    if (mm_read_banner(f, &matcode) != 0)
    {
        printf("mm_read_unsymetric: Could not process Matrix Market banner ");
        printf(" in file [%s]\n", fname);
        return -1;
    }

    if (!(mm_is_real(matcode) && mm_is_matrix(matcode) &&
          mm_is_sparse(matcode)))
    {
        fprintf(stderr, "Sorry, this application does not support ");
        fprintf(stderr, "Market Market type: [%s]\n",
                mm_typecode_to_str(matcode));
        return -1;
    }

    /* find out size of sparse matrix: M, N, nz .... */

    if (mm_read_mtx_crd_size(f, &M, &N, &nz) != 0)
    {
        fprintf(stderr, "read_unsymmetric_sparse(): could not parse matrix size.\n");
        return -1;
    }

    *M_ = M;
    *N_ = N;
    *nz_ = nz;

    /* reseve memory for matrices */

    I = (int *)malloc(nz * sizeof(int));
    J = (int *)malloc(nz * sizeof(int));
    val = (double *)malloc(nz * sizeof(double));

    *val_ = val;
    *I_ = I;
    *J_ = J;

    /* NOTE: when reading in doubles, ANSI C requires the use of the "l"  */
    /*   specifier as in "%lg", "%lf", "%le", otherwise errors will occur */
    /*  (ANSI C X3.159-1989, Sec. 4.9.6.2, p. 136 lines 13-15)            */

    for (i = 0; i < nz; i++)
    {
        fscanf(f, "%d %d %lg\n", &I[i], &J[i], &val[i]);
        I[i]--; /* adjust from 1-based to 0-based */
        J[i]--;
    }
    fclose(f);

    return 0;
}

int mm_is_valid(MM_typecode matcode)
{
    if (!mm_is_matrix(matcode))
        return 0;
    if (mm_is_dense(matcode) && mm_is_pattern(matcode))
        return 0;
    if (mm_is_real(matcode) && mm_is_hermitian(matcode))
        return 0;
    if (mm_is_pattern(matcode) && (mm_is_hermitian(matcode) ||
                                   mm_is_skew(matcode)))
        return 0;
    return 1;
}

int mm_write_mtx_crd_size(FILE *f, int M, int N, int nz)
{
    if (fprintf(f, "%d %d %d\n", M, N, nz) != 3)
        return MM_COULD_NOT_WRITE_FILE;
    else
        return 0;
}

int mm_read_mtx_array_size(FILE *f, int *M, int *N)
{
    char line[MM_MAX_LINE_LENGTH];
    int num_items_read;
    /* set return null parameter values, in case we exit with errors */
    *M = *N = 0;

    /* now continue scanning until you reach the end-of-comments */
    do
    {
        if (fgets(line, MM_MAX_LINE_LENGTH, f) == NULL)
            return MM_PREMATURE_EOF;
    } while (line[0] == '%');

    /* line[] is either blank or has M,N, nz */
    if (sscanf(line, "%d %d", M, N) == 2)
        return 0;

    else /* we have a blank line */
        do
        {
            num_items_read = fscanf(f, "%d %d", M, N);
            if (num_items_read == EOF)
                return MM_PREMATURE_EOF;
        } while (num_items_read != 2);

    return 0;
}

int mm_write_mtx_array_size(FILE *f, int M, int N)
{
    if (fprintf(f, "%d %d\n", M, N) != 2)
        return MM_COULD_NOT_WRITE_FILE;
    else
        return 0;
}

/*-------------------------------------------------------------------------*/

/******************************************************************/
/* use when I[], J[], and val[]J, and val[] are already allocated */
/******************************************************************/

int mm_read_mtx_crd_data(FILE *f, int M, int N, int nz, int I[], int J[],
                         double val[], MM_typecode matcode)
{
    int i;
    if (mm_is_complex(matcode))
    {
        for (i = 0; i < nz; i++)
            if (fscanf(f, "%d %d %lg %lg", &I[i], &J[i], &val[2 * i], &val[2 * i + 1]) != 4)
                return MM_PREMATURE_EOF;
    }
    else if (mm_is_real(matcode))
    {
        for (i = 0; i < nz; i++)
        {
            if (fscanf(f, "%d %d %lg\n", &I[i], &J[i], &val[i]) != 3)
                return MM_PREMATURE_EOF;
        }
    }

    else if (mm_is_pattern(matcode))
    {
        for (i = 0; i < nz; i++)
            if (fscanf(f, "%d %d", &I[i], &J[i]) != 2)
                return MM_PREMATURE_EOF;
    }
    else
        return MM_UNSUPPORTED_TYPE;

    return 0;
}

int mm_read_mtx_crd_entry(FILE *f, int *I, int *J,
                          double *real, double *imag, MM_typecode matcode)
{
    if (mm_is_complex(matcode))
    {
        if (fscanf(f, "%d %d %lg %lg", I, J, real, imag) != 4)
            return MM_PREMATURE_EOF;
    }
    else if (mm_is_real(matcode))
    {
        if (fscanf(f, "%d %d %lg\n", I, J, real) != 3)
            return MM_PREMATURE_EOF;
    }

    else if (mm_is_pattern(matcode))
    {
        if (fscanf(f, "%d %d", I, J) != 2)
            return MM_PREMATURE_EOF;
    }
    else
        return MM_UNSUPPORTED_TYPE;

    return 0;
}

/************************************************************************
mm_read_mtx_crd()  fills M, N, nz, array of values, and return
type code, e.g. 'MCRS'

if matrix is complex, values[] is of size 2*nz,
 (nz pairs of real/imaginary values)
************************************************************************/

int mm_read_mtx_crd(char *fname, int *M, int *N, int *nz, int **I, int **J,
                    double **val, MM_typecode *matcode)
{
    int ret_code;
    FILE *f;

    if (strcmp(fname, "stdin") == 0)
        f = stdin;
    else if ((f = fopen(fname, "r")) == NULL)
        return MM_COULD_NOT_READ_FILE;

    if ((ret_code = mm_read_banner(f, matcode)) != 0)
        return ret_code;

    if (!(mm_is_valid(*matcode) && mm_is_sparse(*matcode) &&
          mm_is_matrix(*matcode)))
        return MM_UNSUPPORTED_TYPE;

    if ((ret_code = mm_read_mtx_crd_size(f, M, N, nz)) != 0)
        return ret_code;

    *I = (int *)malloc(*nz * sizeof(int));
    *J = (int *)malloc(*nz * sizeof(int));
    *val = NULL;

    if (mm_is_complex(*matcode))
    {
        *val = (double *)malloc(*nz * 2 * sizeof(double));
        ret_code = mm_read_mtx_crd_data(f, *M, *N, *nz, *I, *J, *val,
                                        *matcode);
        if (ret_code != 0)
            return ret_code;
    }
    else if (mm_is_real(*matcode))
    {
        *val = (double *)malloc(*nz * sizeof(double));
        ret_code = mm_read_mtx_crd_data(f, *M, *N, *nz, *I, *J, *val,
                                        *matcode);
        if (ret_code != 0)
            return ret_code;
    }

    else if (mm_is_pattern(*matcode))
    {
        ret_code = mm_read_mtx_crd_data(f, *M, *N, *nz, *I, *J, *val,
                                        *matcode);
        if (ret_code != 0)
            return ret_code;
    }

    if (f != stdin)
        fclose(f);
    return 0;
}

int mm_write_banner(FILE *f, MM_typecode matcode)
{
    char *str = mm_typecode_to_str(matcode);
    int ret_code;

    ret_code = fprintf(f, "%s %s\n", MatrixMarketBanner, str);
    free(str);
    if (ret_code != 2)
        return MM_COULD_NOT_WRITE_FILE;
    else
        return 0;
}

int mm_write_mtx_crd(char fname[], int M, int N, int nz, int I[], int J[],
                     double val[], MM_typecode matcode)
{
    FILE *f;
    int i;

    if (strcmp(fname, "stdout") == 0)
        f = stdout;
    else if ((f = fopen(fname, "w")) == NULL)
        return MM_COULD_NOT_WRITE_FILE;

    /* print banner followed by typecode */
    fprintf(f, "%s ", MatrixMarketBanner);
    fprintf(f, "%s\n", mm_typecode_to_str(matcode));

    /* print matrix sizes and nonzeros */
    fprintf(f, "%d %d %d\n", M, N, nz);

    /* print values */
    if (mm_is_pattern(matcode))
        for (i = 0; i < nz; i++)
            fprintf(f, "%d %d\n", I[i], J[i]);
    else if (mm_is_real(matcode))
        for (i = 0; i < nz; i++)
            fprintf(f, "%d %d %20.16g\n", I[i], J[i], val[i]);
    else if (mm_is_complex(matcode))
        for (i = 0; i < nz; i++)
            fprintf(f, "%d %d %20.16g %20.16g\n", I[i], J[i], val[2 * i],
                    val[2 * i + 1]);
    else
    {
        if (f != stdout)
            fclose(f);
        return MM_UNSUPPORTED_TYPE;
    }

    if (f != stdout)
        fclose(f);

    return 0;
}

/**
*  Create a new copy of a string s.  mm_strdup() is a common routine, but
*  not part of ANSI C, so it is included here.  Used by mm_typecode_to_str().
*
*/
char *mm_strdup(const char *s)
{
    int len = strlen(s);
    char *s2 = (char *)malloc((len + 1) * sizeof(char));
    return strcpy(s2, s);
}

// GPU implementation of matrix_vector product: see if you can use
// one thread per row. You'll need to get the addressing right!
// each block of rows.
__global__ void gpuMatrixVector(int rows, int cols, const float *A,
                                const float *x, float *y)
{
    int tr = threadIdx.x;
    int row = blockIdx.x * blockDim.x + tr;
    if (row < rows)
    {
        // Starting address of indexing within matrix A
        int idxm = row * cols;
        float t = 0.0;
        for (int ic = 0; ic < cols; ic++)
        {
            t += A[idxm] * x[ic];
            idxm++;
        }
        y[row] = t;
    }
}

// Simple CPU implementation of matrix-vector product
void MatrixVectorCSR(int M, const int *IRP, const int *JA, const double *AS, const double *x, double *y)
{
    int i, j;
    double t;
    for (i = 0; i < M; ++i)
    {
        t = 0.0;
        for (j = IRP[i]; j < IRP[i + 1]; ++j)
        {
            t += AS[j] * x[JA[j]];
        }
        y[i] = t;
    }
}

__global__ void MatrixVectorCSRParallel(int M, const int *IRP, const int *JA, const double *AS, const double *x, double *y)
{
    int tr = threadIdx.x;
    int m = blockIdx.x * blockDim.x + tr;
    if (m < M)
    {
        double sum = 0.0;
        for (j = IRP[m]; j < IRP[m + 1]; ++j)
        {
            sum += AS[j] * x[JA[j]]
        }
        y[m] = sum;
    }
}

int main(int argc, char *argv[])
{
    int ret_code;
    MM_typecode matcode;
    FILE *f;
    int M, N, nz;
    int i, *I, *J;
    double *val;
    bool isCsrFormat;

    if (argc < 3)
    {
        fprintf(stderr, "Usage: %s [martix-market-filename] [1 for CSR, 2 for Ellpack]\n", argv[0]);
        exit(1);
    }
    else
    {
        if ((f = fopen(argv[1], "r")) == NULL)
            exit(1);
        if (strcmp(argv[2], "1") == 0)
        {
            isCsrFormat = true;
        }
        else if (strcmp(argv[2], "2") == 0)
        {
            isCsrFormat = false;
        }
        else
        {
            printf("Second argument should be 1 for CSR or 2 for ELLPACK\n");
            exit(1);
        }
    }

    if (mm_read_banner(f, &matcode) != 0)
    {
        printf("Could not process Matrix Market banner.\n");
        exit(1);
    }

    /*  This is how one can screen matrix types if their application */
    /*  only supports a subset of the Matrix Market data types.      */

    if (mm_is_complex(matcode) && mm_is_matrix(matcode) &&
        mm_is_sparse(matcode))
    {
        printf("Sorry, this application does not support ");
        printf("Market Market type: [%s]\n", mm_typecode_to_str(matcode));
        exit(1);
    }

    /* find out size of sparse matrix .... */

    if ((ret_code = mm_read_mtx_crd_size(f, &M, &N, &nz)) != 0)
        exit(1);

    /* reseve memory for matrices */

    I = (int *)malloc(nz * sizeof(int));
    J = (int *)malloc(nz * sizeof(int));
    val = (double *)malloc(nz * sizeof(double));

    /* NOTE: when reading in doubles, ANSI C requires the use of the "l"  */
    /*   specifier as in "%lg", "%lf", "%le", otherwise errors will occur */
    /*  (ANSI C X3.159-1989, Sec. 4.9.6.2, p. 136 lines 13-15)            */

    for (i = 0; i < nz; i++)
    {
        fscanf(f, "%d %d %lg\n", &I[i], &J[i], &val[i]);
        I[i]--; /* adjust from 1-based to 0-based */
        J[i]--;
    }

    if (f != stdin)
        fclose(f);

    /************************/
    /* now write out matrix */
    /************************/

    mm_write_banner(stdout, matcode);
    mm_write_mtx_crd_size(stdout, M, N, nz);
    for (i = 0; i < nz; i++)
        fprintf(stdout, "%d %d %20.19g\n", I[i], J[i], val[i]);
    printf("Columns : %d, rows : %d, non-zeros : %d\n\n", M, N, nz);

    // CREATING VECTORS
    double *x = (double *)malloc(sizeof(double) * M);
    double *y = (double *)malloc(sizeof(double) * M);

    for (i = 0; i < M; i++)
    {
        x[i] = 100.0f * ((double)rand()) / RAND_MAX;
    }

    if (isCsrFormat == true)
    {
        /*************************/
        /* CSR FORMAT CALCULATION*/
        /*************************/
        int *IRP = (int *)malloc((M + 1) * sizeof(int));
        // ASSUMING MATLAB FIRST COLUMN INDEXING
        IRP[0] = 1;
        int index = 0;
        int local_row_nz = 1;
        for (i = 0; i < nz; i++)
        {
            if (I[i] == I[i + 1])
            {
                local_row_nz++;
            }
            else
            {
                if (index <= M)
                {
                    IRP[index + 1] = IRP[index] + local_row_nz;
                    local_row_nz = 1;
                    index++;
                }
            }
        }

        // ----------------------- Host memory initialisation ----------------------- //

        double *h_y_d = new double[M];
        std::cout << "Matrix-vector product: single thread per row version " << std::endl;
        std::cout << "Test case: " << M << " x " << M << std::endl;

        // ---------------------- Device memory initialisation ---------------------- //
        //  Allocate memory space on the device.

        int *d_IRP, *d_J;
        double *d_val, *d_x, *d_y;

        checkCudaErrors(hipMalloc((void **)&d_IRP, (M+1) * sizeof(int)));
        checkCudaErrors(hipMalloc((void **)&d_J, nz * sizeof(int)));
        checkCudaErrors(hipMalloc((void **)&d_val, nz * sizeof(double)));
        checkCudaErrors(hipMalloc((void **)&d_x, M * sizeof(double)));
        checkCudaErrors(hipMalloc((void **)&d_y, M * sizeof(double)));

        // Copy matrices from the host (CPU) to the device (GPU).
        
        checkCudaErrors(hipMemcpy(d_IRP, IRP, (M+1) * sizeof(int), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_J, J, nz * sizeof(int), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_val, val, nz * sizeof(double), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_x, x, M * sizeof(double), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_y, y, M * sizeof(double), hipMemcpyHostToDevice));


        // ------------------------ Calculations on the CPU ------------------------- //
        float flopcnt = 2.e-6 * nrows * ncols;

        // Create the CUDA SDK timer.
        StopWatchInterface *timer = 0;
        sdkCreateTimer(&timer);

        timer->start();
        // CpuMatrixVector(nrows, ncols, h_A, h_x, h_y);
        MatrixVectorCSR(M, IRP, J, val, x, y);

        timer->stop();
        float cpuflops = flopcnt / timer->getTime();
        std::cout << "  CPU time: " << timer->getTime() << " ms."
                  << " GFLOPS " << cpuflops << std::endl;

        // ------------------------ Calculations on the GPU ------------------------- //

        // Calculate the dimension of the grid of blocks (1D) necessary to cover
        // all rows.
        const dim3 GRID_DIM((nrows - 1 + BLOCK_DIM.x) / BLOCK_DIM.x, 1);

        timer->reset();
        timer->start();
        MatrixVectorCSRParallel<<<GRID_DIM, BLOCK_DIM>>>(M, d_IRP, d_J, d_val, d_x, d_y);
        checkCudaErrors(hipDeviceSynchronize());

        timer->stop();
        float gpuflops = flopcnt / timer->getTime();
        std::cout << "  GPU time: " << timer->getTime() << " ms."
                  << " GFLOPS " << gpuflops << std::endl;

        // Download the resulting vector d_y from the device and store it in h_y_d.
        checkCudaErrors(hipMemcpy(h_y_d, d_y, nrows * sizeof(float), hipMemcpyDeviceToHost));

        // Now let's check if the results are the same.
        float reldiff = 0.0f;
        float diff = 0.0f;

        for (int i = 0; i < M; ++i)
        {
            float maxabs = std::max(std::abs(y[i]), std::abs(h_y_d[i]));
            if (maxabs == 0.0)
                maxabs = 1.0;
            reldiff = std::max(reldiff, std::abs(y[i] - h_y_d[i]) / maxabs);
            diff = std::max(diff, std::abs(y[i] - h_y_d[i]));
        }
        std::cout << "Max diff = " << diff << "  Max rel diff = " << reldiff << std::endl;
        // Rel diff should be as close as possible to unit roundoff; float
        // corresponds to IEEE single precision, so unit roundoff is
        // 1.19e-07
        //

        // ------------------------------- Cleaning up ------------------------------ //

        delete timer;

        checkCudaErrors(hipFree(d_A));
        checkCudaErrors(hipFree(d_x));
        checkCudaErrors(hipFree(d_y));

        delete[] h_A;
        delete[] h_x;
        delete[] h_y;
        delete[] h_y_d;
    }
    return 0;
}